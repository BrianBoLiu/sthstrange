#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <ctype.h>
#include <time.h>
#include <math.h>
#include <signal.h>

#include "sha1.h"

#define THREAD_LOOPS 100000

int sigintfired = 0;
void handle_sigint(int signo) {
    if (signo == SIGINT)
        sigintfired = 1;
}

double current_millis() {
	struct timespec spec;
	clock_gettime(CLOCK_REALTIME, &spec);
	
	double sec = spec.tv_sec;
	double ms = round(spec.tv_nsec / 1.0e6);
	return ms + 1000.0 * sec;
}

// Updates 20-byte SHA-1 record in 'hash' for 'num_blocks' consequtive 64-byte blocks
extern "C"
void sha1_update_intel(int *hash, unsigned char* input, size_t num_blocks );

typedef struct {
	char success;
	char nonce[17];
} result_t;

__device__ int compare(hash_digest_t *hash, unsigned char *target) {
	uint32_t *tmp = (uint32_t *)hash;
	#pragma unroll 5
	for (unsigned i = 0; i < 5; i++) {
		tmp[i] = swap(tmp[i]);
	}
	unsigned char *h = (unsigned char *)hash;
	#pragma unroll 20
	for (int i = 0; i < 20; i++) {
		if (h[i] == target[i]) continue;
		if (h[i] < target[i]) return 1;
		return 0;
	}
	return 0;
}

// Get as many threads as we like to simultaneously SHA1 data, store in results.
// data should already be padded etc.
__global__ void parallel_hash(const unsigned char *lastblock, const unsigned char *target, result_t *results, hash_digest_t hashinit) {
	// Copy the last block into local memory.
	unsigned char blk[64];
	memcpy(blk, lastblock, 64);
	
	// Copy the target into local memory
	unsigned char trg[20];
	memcpy(trg, target, 20);
	
	// The nonce
	unsigned char *nonce = &blk[64-25];
	
	// Take the first two chars to uniquely identify ourselves
	nonce[0] = '0' + blockIdx.x / 64;
	nonce[1] = '0' + blockIdx.x % 64;
	nonce[2] = '0' + threadIdx.x / 64;
	nonce[3] = '0' + threadIdx.x % 64;

	uint32_t w[16];
	hash_digest_t h;
	char success = 0;
	for (int i = 0; i < THREAD_LOOPS; i++) {
		int pos = i;
		// 64^4 is approx 16e6 so we should be good
		nonce[4] = '0' + (pos&0x3f); pos >>= 6;
		nonce[5] = '0' + (pos&0x3f); pos >>= 6;
		nonce[6] = '0' + (pos&0x3f); pos >>= 6;
		nonce[7] = '0' + (pos&0x3f); pos >>= 6;
		h = hashinit;
		computeSHA1Block(blk, w, &h);
		if (compare(&h, trg)) {
			success = 1;
			break;
		}
	}
	
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	results[idx].success = success;
	memcpy(results[idx].nonce, nonce, 16);
	results[idx].nonce[16] = '\0';
}

int get_difficulty(char *hex, unsigned char *target) {
	if (strlen(hex) > 40)
		return 0;
	for (int i = 0; i < strlen(hex); i++) {
		char c = hex[i];
		if (!isxdigit(c))
			return 0;
		int num = ('0' <= c && c <= '9') ? c - '0' : ('a' <= c && c <= 'f') ? c - 'a' + 10 : c - 'A' + 10;
		if (i % 2 == 0)
			target[i/2] |= num << 4;
		else
			target[i/2] |= num;
	}
	return 1;
}

int main(int argc, char *argv[]) {
	// Must be used with a difficulty or "bench"
	if (argc != 2) {
		printf("Usage: %s <difficulty in hex | bench>\n", argv[0]);
		return 1;
	}
	
	// Check for benchmark flag
	int bench = 0;
	if (strcmp(argv[1], "bench") == 0)
		bench = 1;
	
	// Read difficulty
	unsigned char target[20] = {0};
	if (!bench) {
		// In benchmarking mode, all zeros.
		if (!get_difficulty(argv[1], target)) {
			printf("Difficulty must be a hex string no longer than 40 chars.\n");
			return 1;
		}
	}
	
	// DEBUG difficulty
	// for (int i = 0; i < 20; i++) printf("%d ", target[i]); putchar('\n');
	
	// Read data
	unsigned char data[1024] = {0};
	int n, nread = 0;
	
	if (bench) {
		// In benchmarking mode, prefill the data.
		nread = 295;
		memset(data, 'A', nread);
	} else {
		while ((n = fread(data, 1, sizeof(data) - nread, stdin)) != 0)
			nread += n;
	}
	
	if (nread % 64 != 39) {
		printf("nread mod 64 = %d, wanted 39\n", nread % 64);
		return 1;
	}
	
	// Place the nonce
	unsigned char *nonce = &data[nread];
	memset(nonce, '0', 16);
	nread += 16;
	
	// Finish the SHA1 padding
	int nbits = 8*nread;
	int pos = nread;
	data[pos++] = 0x80;
	pos += 8;
	data[pos-1] = nbits & 0xff;
	data[pos-2] = (nbits & 0xff00) >> 8;
	data[pos-3] = (nbits & 0xff0000) >> 16;
	
	// DEBUG data
	// for (int i = 0; i < pos; i++) printf("%d ", data[i]); putchar('\n');
	
	// Hash most of it
	hash_digest_t hashinit = SHA1_DIGEST_INIT;
	if (pos/64 - 1 > 0)
		sha1_update_intel((int*)&hashinit, data, pos/64 - 1);
	
	int nblocks = 120;
	int nthreads = 64;
	int nprocs = nblocks * nthreads;
	
	dim3 dimGrid(nblocks);
	dim3 dimBlock(nthreads);
	
	// Allocate some device memory for threads to store their results.
	result_t *results_dev;
	hipMalloc(&results_dev, nprocs * sizeof(result_t));
	result_t *results = (result_t *) malloc(sizeof(result_t) * nprocs);
	
	// Pass the last data block to the device.
	unsigned char *data_dev;
	hipMalloc(&data_dev, sizeof(data));
	unsigned char *lastblock_dev = &data_dev[pos-64];

	// Pass the target to the device.
	unsigned char *target_dev;
	hipMalloc(&target_dev, sizeof(target));
	hipMemcpy(target_dev, target, sizeof(target), hipMemcpyHostToDevice);

	int runs = 1;
	result_t *answer = NULL;
    signal(SIGINT, handle_sigint);
	double start = current_millis();
	for (;answer == NULL;runs++) {
        // Check for early exit
        if (sigintfired) {
            printf("SIGINT Receieved, exiting.\n");
            return 1;
        }
		// Modify the nonce
		int pos = runs;
		// 64^4 is approx 16e6 so we should be good
		nonce[10] = '0' + (pos&0x3f); pos >>= 6;
		nonce[11] = '0' + (pos&0x3f); pos >>= 6;
		nonce[12] = '0' + (pos&0x3f); pos >>= 6;
		nonce[13] = '0' + (pos&0x3f); pos >>= 6;
		nonce[14] = '0' + (pos&0x3f); pos >>= 6;
		nonce[15] = '0' + (pos&0x3f); pos >>= 6;

        /*
        nonce[16] = '\0';
        printf("%s\n", nonce);
        nonce[16] = 0x80;
        */

		// Copy the new data to the GPU
		hipMemcpy(data_dev, data, sizeof(data), hipMemcpyHostToDevice);

		// Launch the kernel.
		parallel_hash<<<dimGrid, dimBlock>>>(lastblock_dev, target_dev, results_dev, hashinit);

		hipDeviceSynchronize();
		hipError_t error = hipGetLastError();
		if(error!=hipSuccess) {
			fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
			exit(-1);
		}
	
		// Copy the hashes back to main memory.
		hipMemcpy(results, results_dev, sizeof(result_t) * nprocs, hipMemcpyDeviceToHost);
		
		// Did we find anything? (Virtually no time impact)
		for (int i = 0; i < nblocks * nthreads; i++) {
			if (results[i].success) {
				answer = &results[i];
				break;
			}
		}
		
		// If we're benchmarking, quit after 10 runs.
		if (bench && runs == 10)
			break;
	}
	
	if (bench) {
		printf("Completed %d runs\n", runs);
		double end = current_millis();
		double secs = (end - start) / 1000.0;
		double hashes = (double)nprocs * (double)runs * (double)THREAD_LOOPS;
		printf("%.2f mhash/sec\n", hashes / secs / 1e6);
		return 0;
	}
	
	memcpy(nonce, answer->nonce, 16);
	fwrite(data, 1, nread, stdout);
	return 0;
}
	
	
	
	
